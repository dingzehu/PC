
#include <hip/hip_runtime.h>
#include <stdio.h>

static const int blockSize = 1024;

__global__ void sum_d( int N, float *arr ) {
  int idx = threadIdx.x;
  int sum = 0;
  for (int i = idx; i < N; i += blockSize)
    sum += arr[i];
  arr[idx] = sum;
  __syncthreads();
  for (int size = blockSize/2; size>0; size/=2) {
    if (idx<size) arr[idx] += arr[idx+size];
    __syncthreads();
  }
}

int main(void) {
  int N = 1024*1024, i;
  float arr[N], sum = 0;
  float *arr_d;

   hipMalloc( &arr_d, sizeof(float)*N);

   for (i=0; i<N; i++) {
      arr[i] = i;
      sum += i;
   }
   
   hipMemcpy(arr_d, arr, sizeof(float)*N, hipMemcpyHostToDevice);

   sum_d<<<1,blockSize>>>(N,arr_d);

   hipMemcpy(arr, arr_d , sizeof(float), hipMemcpyDeviceToHost);

   printf( "sum = %f\n", sum );
   printf( "sum from d = %f\n", arr[0] );

   hipFree(arr_d);
}

