
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_d( int N, float *a, float *b, float *c ) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < N) c[i] = a[i] + b[i];

}

int main(void) {
   int N = 128, i;
   float *a, *b, *c;
   float *a_d, *b_d, *c_d;

   a = (float *)malloc( sizeof(float)*N );
   b = (float *)malloc( sizeof(float)*N );
   c = (float *)malloc( sizeof(float)*N );

   hipMalloc( &a_d, sizeof(float)*N);
   hipMalloc( &b_d, sizeof(float)*N);
   hipMalloc( &c_d, sizeof(float)*N);

   for (i=0; i<N; i++) {
      a[i] = i;
      b[i] = 2*i;
   }

   hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
   hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);
   add_d<<<1,N>>>(N,a_d, b_d, c_d);
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   float elapsedTime;
   hipEventElapsedTime(&elapsedTime, start, stop);

   hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

   for(i=0; i<N; i++) {
        printf( "%f + %f = %f\n", a[i], b[i], c[i] );
  }
   printf("%f\n",elapsedTime);
   hipFree(a_d); hipFree(b_d); hipFree(c_d);
}

